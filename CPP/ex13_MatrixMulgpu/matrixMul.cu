

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define THREAD_SIZE	(2)

#define MATRIX_A_X		(2 * THREAD_SIZE)
#define MATRIX_A_Y		(1 * THREAD_SIZE)

#define MATRIX_B_X		(1 * THREAD_SIZE)
#define MATRIX_B_Y		(2 * THREAD_SIZE)


void matrixMul_RefC(int A[MATRIX_A_Y][MATRIX_A_X], int B[MATRIX_B_Y][MATRIX_B_X], int C[MATRIX_A_Y][MATRIX_B_X])
{
	//Reset the result buffer
	for (int y = 0; y < MATRIX_A_Y; y++) {
		for (int x = 0; x < MATRIX_B_X; x++)
		{
			C[y][x] = 0;
		}
	}

	//Matrix Multiplication
	for (int z = 0; z < MATRIX_B_X; z++) {
		for (int y = 0; y < MATRIX_A_Y; y++) {
			for (int x = 0; x < MATRIX_A_X; x++) {
				C[z][y] += A[y][x] * B[x][z];
			}
		}
	}

	//Print the result
	printf("Reference Source :: Matrix Multiplication Result (C = A * B): \n");
	for (int y = 0; y < MATRIX_A_Y; y++) {
		for (int x = 0; x < MATRIX_B_X; x++)
		{
			printf("%d ", C[y][x]);
		}
		printf("\n");
	}
}

__global__ void matrixMul_Kernel(int *A, int *B, int *C, int pitchA, int pitchB, int pitchC, int nElem)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	//Matrix Multiplication
	int cSum = 0;
	for (int k = 0; k < nElem; k++) {
		cSum += A[x * pitchA + k] * B[k * pitchB + y];
	}

	C[y * pitchC + x] = cSum;
}


void matrixMul_CUDA(int A[MATRIX_A_Y][MATRIX_A_X], int B[MATRIX_B_Y][MATRIX_B_X], int C[MATRIX_A_Y][MATRIX_B_X])
{
	hipError_t error = hipSuccess;
	hipDeviceProp_t deviceProp;	

	int devId = 0;
	if (hipSuccess != (error = hipGetDevice(&devId))) {
		printf("hipGetDevice CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	if (hipSuccess != (error = hipGetDeviceProperties(&deviceProp, devId))) {
		printf("hipGetDeviceProperties CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	printf("\nGPU Device %d: \"%s\" with compute capability %d.%d\n", devId, deviceProp.name, deviceProp.major, deviceProp.minor);

	int *d_A = NULL;
	int *d_B = NULL;
	int *d_C = NULL;

	if (hipSuccess != (error = hipMalloc((void**)&d_A, (sizeof(int) * MATRIX_A_Y * MATRIX_A_X)))) {
		printf("hipMalloc CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	if (hipSuccess != (error = hipMalloc((void**)&d_B, (sizeof(int) * MATRIX_B_Y * MATRIX_B_X)))) {
		printf("hipMalloc CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	if (hipSuccess != (error = hipMalloc((void**)&d_C, (sizeof(int) * MATRIX_A_Y * MATRIX_B_X)))) {
		printf("hipMalloc CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	if (hipSuccess != (error = hipMemcpy((void*)d_A, (void*)A, (sizeof(int) * MATRIX_A_Y * MATRIX_A_X), hipMemcpyHostToDevice))) {
		printf("hipMemcpy CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	if (hipSuccess != (error = hipMemcpy((void*)d_B, (void*)B, (sizeof(int) * MATRIX_B_Y * MATRIX_B_X), hipMemcpyHostToDevice))) {
		printf("hipMemcpy CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	dim3 thread(THREAD_SIZE, THREAD_SIZE);
	dim3 grid((MATRIX_A_Y / THREAD_SIZE), (MATRIX_B_X / THREAD_SIZE));

	int nElement = MATRIX_A_X;
	int pitchA = MATRIX_A_X;
	int pitchB = MATRIX_B_X;

	int pitchC  = MATRIX_A_Y;

	matrixMul_Kernel<<<grid, thread>>>(d_A, d_B, d_C, pitchA, pitchB, pitchC, nElement);

	if (hipSuccess != (error = hipDeviceSynchronize())) {
		printf("hipDeviceSynchronize CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	//Reset the result buffer
	for (int y = 0; y < MATRIX_A_Y; y++) {
		for (int x = 0; x < MATRIX_B_X; x++)
		{
			C[y][x] = 0;
		}
	}
	
	if (hipSuccess != (error = hipMemcpy((void*)C, (void*)d_C, (sizeof(int) * MATRIX_A_Y * MATRIX_B_X), hipMemcpyDeviceToHost))) {
		printf("hipMemcpy CUDA Error: %s (code: %d)\n", hipGetErrorString(error), error);
	}

	//Print the result
	printf("CUDA Source :: Matrix Multiplication Result (C = A * B): \n");
	for (int y = 0; y < MATRIX_A_Y; y++) {
		for (int x = 0; x < MATRIX_B_X; x++)
		{
			printf("%d ", C[y][x]);
		}
		printf("\n");
	}
}

int main()
{
	int A[MATRIX_A_Y][MATRIX_A_X];
	int B[MATRIX_B_Y][MATRIX_B_X];
	int C[MATRIX_A_Y][MATRIX_B_X];

	//Filling the A and B matrix

	for (int y = 0; y < MATRIX_A_Y; y++) {
		for (int x = 0; x < MATRIX_A_X; x++)
		{
			A[y][x] = y * MATRIX_A_X + x + 1;
		}
	}

	for (int y = 0; y < MATRIX_B_Y; y++) {
		for (int x = 0; x < MATRIX_B_X; x++)
		{
			B[y][x] = y * MATRIX_B_X + x + 1;
		}
	}

	printf("A = \n");
	for (int y = 0; y < MATRIX_A_Y; y++) {
		for (int x = 0; x < MATRIX_A_X; x++)
		{
			printf("%d ", A[y][x]);
		}
		printf("\n");
	}

	printf("B = \n");
	for (int y = 0; y < MATRIX_B_Y; y++) {
		for (int x = 0; x < MATRIX_B_X; x++)
		{
			printf("%d ", B[y][x]);
		}
		printf("\n");
	}

	//Call Reference Source
	matrixMul_RefC(A, B, C);

	//Call CUDA kernel
	matrixMul_CUDA(A, B, C);
	

	return 0;
}